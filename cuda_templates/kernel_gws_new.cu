#include "hip/hip_runtime.h"
/* PyCOOL v. 0.997300203937
Copyright (C) 2011/04 Jani Sainio <jani.sainio@utu.fi>
Distributed under the terms of the GNU General Public License
http://www.gnu.org/licenses/old-licenses/gpl-2.0.txt

Please cite arXiv:
if you use this code in your research.
See also http://www.physics.utu.fi/tiedostot/theory/particlecosmology/pycool/ .

Part of this code adapted from CUDAEASY
http://www.physics.utu.fi/tiedostot/theory/particlecosmology/cudaeasy/
(See http://arxiv.org/abs/0911.5692 for more information.),
LATTICEEASY
http://www.science.smith.edu/departments/Physics/fstaff/gfelder/latticeeasy/ ,
DEFROST http://www.sfu.ca/physics/cosmology/defrost .
(See http://arxiv.org/abs/0809.4904 for more information.),
Nvidia SDK FDTD3dGPU kernel
(See http://developer.nvidia.com/gpu-computing-sdk .) 
and from HLattice HLattice.
(See http://arxiv.org/abs/1102.0227 for more information.)
*/


__constant__ {{ type_name_c }} c2_coeff[5];
__constant__ {{ type_name_c }} gw_coeff[1];

////////////////////////////////////////////////////////////////////////////////
// Gravitational wave kernels
////////////////////////////////////////////////////////////////////////////////

__device__ double atomicAdd(double* address, double val)
// Double precision atomic add function
{
    double old = *address, assumed;
    do {
        assumed = old;
        old = __longlong_as_double(
                     atomicCAS((unsigned long long int*)address,
                                __double_as_longlong(assumed),
                                __double_as_longlong(val + assumed)));
        } while (assumed != old);
     return old;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Calculate the laplacian of one component of the u_ij tensor and evolve the canonical momentum of the tensor field
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void kernelU3_{{ tensor_ij_c }}({{ type_name_c }} *u11_m, {{ type_name_c }} *u12_m, {{ type_name_c }} *u22_m, {{ type_name_c }} *u13_m, {{ type_name_c }} *u23_m, {{ type_name_c }} *u33_m, {{ type_name_c }} *piu11_m, {{ type_name_c }} *piu12_m, {{ type_name_c }} *piu22_m, {{ type_name_c }} *piu13_m, {{ type_name_c }} *piu23_m, {{ type_name_c }} *piu33_m)

// This function calculates \nabla^2 u_{ij} for the u_{ij} components
// and evolves pi_u_{ij}
{

    {% set radius2 = 2*radius_c %}
    {% set radiusp1 = radius_c + 1%}{% set radiusm1 = radius_c - 1%}{% set radiusp2 = radius_c + 2%}{% set radiusm2 = radius_c - 2%}
    {% set radiusp3 = radius_c + 3%}{% set radiusm3 = radius_c - 3%}{% set radiusp4 = radius_c + 4%}{% set radiusm4 = radius_c - 4%}

    {% set blockIdx01 = DIM_X_c - radius_c %}
    {% set blockIdx02 = radius_c%}
    {% set blockIdx11 = DIM_X_c - block_x_c%}
    {% set blockIdx12 = block_x_c%}

    {% set blockIdy01 = DIM_X_c*(DIM_Y_c - radius_c) %}
    {% set blockIdy02 = DIM_X_c*radius_c %}
    {% set blockIdy11 = DIM_X_c*(DIM_Y_c - block_y_c) %}
    {% set blockIdy12 = DIM_X_c*(block_y_c) %}

    {% set gridx1 =  grid_x_c - 1 %}
    {% set gridy1 =  grid_y_c - 1 %}

    {% set blockx1 =  block_x_c + radius_c %}
    {% set blocky1 =  block_y_c + radius_c %}

    {% set down_idx =  stride_c*(DIM_Z_c - 1)%}

    // Shared data used in the calculation of the Laplacian of the field f
    __shared__ {{ type_name_c }} s_data[{{ block_y_c }} + {{ radius2 }}][{{ block_x_c }} + {{ radius2 }}];

    // Thread ids
    // in_idx is used to load data into the top of the stencil
    // out_idx is used to load data into the shared memory
    volatile unsigned int out_idx = {{ DIM_X_c }}*(blockIdx.y*blockDim.y + threadIdx.y) + blockIdx.x*blockDim.x + threadIdx.x;
    volatile unsigned int in_idx = out_idx + {{ down_idx }};
    volatile unsigned int i,j;

    //volatile unsigned int stride_z = {{ stride_c }};

    //{{ type_name_c }} f{{ field_i_c }};
    //{{ type_name_c }} pi{{ field_i_c }};

    {{ type_name_c }} up[{{ radius_c }}];
    {{ type_name_c }} down[{{ radius_c }}];

   {% for i in other_i_c %} {{ type_name_c }} f{{i}};
   {% endfor %}
    {{ type_name_c }} D2u;

    /////////////////////////////////////////
    // load the initial data into shared mem
    // down data from the top of the lattice
    // due to the periodicity of the lattice

    // Down data
    // In a multi-gpu implementation these values could be loaded from a different device
//#pragma unroll {{ radiusp1 }}
    for (j = 0; j < {{ radius_c }}; j++)
    {
        down[j] = u{{ tensor_ij_c }}_m[in_idx];
        in_idx -= {{ stride_c }};
    }

    //  Inner points of shared memory
    s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx];

    // West halo
    if (blockIdx.x == 0){
        // Periodic lattice
        // In a multi-gpu implementation these values could be loaded from a different device
        if (threadIdx.x < {{ radius_c }}){
            s_data[threadIdx.y + {{ radius_c }}][threadIdx.x] = u{{ tensor_ij_c }}_m[out_idx + ({{ blockIdx01 }})];
        }
    }
    else {
        if (threadIdx.x < {{ radius_c }}){
            s_data[threadIdx.y + {{ radius_c }}][threadIdx.x] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdx02 }})];
        }
    }
    // East halo
    if (blockIdx.x == {{ gridx1 }}){
        // Periodic lattice
        // In a multi-gpu implementation these values could be loaded from a different device
        if (threadIdx.x < {{ radius_c }}){
            //sumterm_w[out_idx] = out_idx -({{ blockIdx11 }});
            s_data[threadIdx.y + {{ radius_c }} ][threadIdx.x + {{ blockx1 }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdx11 }})];
        }
    }
    else {
        if (threadIdx.x < {{ radius_c }}){
            s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ blockx1 }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdx12 }}];
        }
    }
    // North halo
    if (blockIdx.y == 0){
        // Periodic lattice
        // In a multi-gpu implementation these values could be loaded from a different device
        if (threadIdx.y < {{ radius_c }}){
            s_data[threadIdx.y][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdy01 }}];
        }
    }
    else {
        if (threadIdx.y < {{ radius_c }}){
            //sumterm_w[out_idx] = out_idx - ({{ blockIdy02 }});
            s_data[threadIdx.y][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdy02 }})];
         }
    }

    // South halo
    if (blockIdx.y == {{ gridy1 }}){
        // Periodic lattice
        // In a multi-gpu implementation these values could be loaded fjennifer lopez heightrom a different device
        if (threadIdx.y < {{ radius_c }}){
            s_data[threadIdx.y + {{ blocky1 }}][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdy11 }})];
        }
    }
    else {
        if (threadIdx.y < {{ radius_c }}){
            s_data[threadIdx.y + {{ blocky1 }}][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdy12 }}];
         }
    }

    // Up data
    // In a multi-gpu implementation these values could be loaded from a different device
    in_idx = out_idx + {{ stride_c }};

//#pragma unroll {{ radius_c }}
    for (j = 0 ; j < {{ radius_c }} ; j++)
    {
        up[j] = u{{ tensor_ij_c }}_m[in_idx];
        in_idx += {{ stride_c }};
    }
    
    __syncthreads();

    /////////////////////////////////////////
    // Calculations

    // Discretized Laplacian operator
    // gw_coeff[0] = dt*a(t)^2/(dx^2)
    // c2_coeff's = laplacian discretization coefficients

    {% if radius_c == 4%}
        D2u = gw_coeff[0]*(c2_coeff[4]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp4 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm4 }}] +
                                       s_data[threadIdx.y + {{ radiusp4 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm4 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm1 }}] + down[{{ radiusm1 }}]) +
                          c2_coeff[3]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp3 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm3 }}] +
                                       s_data[threadIdx.y + {{ radiusp3 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm3 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm2 }}] + down[{{ radiusm2 }}]) +   
                          c2_coeff[2]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp2 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm2 }}] +
                                       s_data[threadIdx.y + {{ radiusp2 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm2 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm3 }}] + down[{{ radiusm3 }}]) +
                          c2_coeff[1]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp1 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm1 }}] +
                                       s_data[threadIdx.y + {{ radiusp1 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm1 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm4 }}] + down[{{ radiusm4 }}]) +
                          c2_coeff[0]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radius_c }}]));

    {% endif %}

    /////////////////////////////////
    //  Evolution step
    /////////////////////////////////

        piu{{ tensor_ij_c }}_m[out_idx] += D2u;


    {% set foo = DIM_Z_c-radius_c %}
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // advance in z direction until z={{ foo }}
    // {{ foo }} <= z < {{ DIM_Z_c }} calculated seperately

//#pragma unroll {{ foo }}
    for(i=1; i<({{ foo }}); i++)
    {
        __syncthreads();

        // Advance the slice (move the thread-front)
//#pragma unroll {{ radiusm1 }}
        for (int j = {{ radiusm1 }} ; j > 0 ; j--)
            down[j] = down[j - 1];

        down[0] = s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radius_c }}];
        s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radius_c }}] = up[0];

//#pragma unroll {{ radiusm1 }}
        for (int j = 0 ; j < {{ radiusm1 }}; j++)
            up[j] = up[j + 1];
        up[{{ radiusm1 }}] = u{{ tensor_ij_c }}_m[in_idx];

        in_idx += {{ stride_c }};
        out_idx += {{ stride_c }};

        // West halo
        if (blockIdx.x == 0){
            // Periodic lattice
            // In a multi-gpu implementation these values could be loaded from a different device
            if (threadIdx.x < {{ radius_c }}){
                s_data[threadIdx.y + {{ radius_c }}][threadIdx.x] = u{{ tensor_ij_c }}_m[out_idx + ({{ blockIdx01 }})];
            }
        }
        else {
            if (threadIdx.x < {{ radius_c }}){
                s_data[threadIdx.y + {{ radius_c }}][threadIdx.x] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdx02 }})];
            }
        }
        // East halo
        if (blockIdx.x == {{ gridx1 }}){
            // Periodic lattice
            // In a multi-gpu implementation these values could be loaded from a different device
            if (threadIdx.x < {{ radius_c }}){
                s_data[threadIdx.y + {{ radius_c }} ][threadIdx.x + {{ blockx1 }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdx11 }})];
            }
        }
        else {
            if (threadIdx.x < {{ radius_c }}){
                s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ blockx1 }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdx12 }}];
            }
        }
        // North halo
        if (blockIdx.y == 0){
            // Periodic lattice
            // In a multi-gpu implementation these values could be loaded from a different device
            if (threadIdx.y < {{ radius_c }}){
                s_data[threadIdx.y][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdy01 }}];
            }
        }
        else {
            if (threadIdx.y < {{ radius_c }}){
                s_data[threadIdx.y][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdy02 }})];
             }
        }
        // South halo
        if (blockIdx.y == {{ gridy1 }}){
            // Periodic lattice
            // In a multi-gpu implementation these values could be loaded from a different device
            if (threadIdx.y < {{ radius_c }}){
                s_data[threadIdx.y + {{ blocky1 }}][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdy11 }})];
            }
        }
        else {
            if (threadIdx.y < {{ radius_c }}){
                s_data[threadIdx.y + {{ blocky1 }}][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdy12 }}];
             }
        }
       
	__syncthreads();

    /////////////////////////////////////////
    // Calculations

    // Discretized Laplacian operator
    // gw_coeff[0] = dt*a(t)^2/(dx^2)
    // c2_coeff's = laplacian discretization coefficients

    {% if radius_c == 4%}
        D2u = gw_coeff[0]*(c2_coeff[4]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp4 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm4 }}] +
                                       s_data[threadIdx.y + {{ radiusp4 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm4 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm1 }}] + down[{{ radiusm1 }}]) +
                          c2_coeff[3]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp3 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm3 }}] +
                                       s_data[threadIdx.y + {{ radiusp3 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm3 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm2 }}] + down[{{ radiusm2 }}]) +   
                          c2_coeff[2]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp2 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm2 }}] +
                                       s_data[threadIdx.y + {{ radiusp2 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm2 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm3 }}] + down[{{ radiusm3 }}]) +
                          c2_coeff[1]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp1 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm1 }}] +
                                       s_data[threadIdx.y + {{ radiusp1 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm1 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm4 }}] + down[{{ radiusm4 }}]) +
                          c2_coeff[0]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radius_c }}]));
    {% endif %}

	/////////////////////////////////
	//  Evolution step
	/////////////////////////////////

          piu{{ tensor_ij_c }}_m[out_idx] += D2u;

    }

    {% set foo = DIM_Z_c-radius_c %}
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // up data now from the bottom of the lattice due to periodicity
    //

    in_idx = {{ DIM_X_c }}*(blockIdx.y*blockDim.y + threadIdx.y) + blockIdx.x*blockDim.x + threadIdx.x;

#pragma unroll {{ radius_c }}
    for(i={{ foo }}; i<({{ DIM_Z_c }}); i++)
    {
        __syncthreads();

        // Advance the slice (move the thread-front)
#pragma unroll {{ radiusm1 }}
        for (j = {{ radiusm1 }} ; j > 0 ; j--)
            down[j] = down[j - 1];

        down[0] = s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radius_c }}];
        s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radius_c }}] = up[0];


#pragma unroll {{ radius_c }}
        for (j = 0 ; j < {{ radiusm1 }}; j++)
            up[j] = up[j + 1];
        up[{{ radiusm1 }}] = u{{ tensor_ij_c }}_m[in_idx];

        in_idx  += {{ stride_c }};
        out_idx  += {{ stride_c }};



        // West halo
        if (blockIdx.x == 0){
            // Periodic lattice
            // In a multi-gpu implementation these values could be loaded from a different device
            if (threadIdx.x < {{ radius_c }}){
                s_data[threadIdx.y + {{ radius_c }}][threadIdx.x] = u{{ tensor_ij_c }}_m[out_idx + ({{ blockIdx01 }})];
            }
        }
        else {
            if (threadIdx.x < {{ radius_c }}){
                s_data[threadIdx.y + {{ radius_c }}][threadIdx.x] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdx02 }})];
            }
        }
        // East halo
        if (blockIdx.x == {{ gridx1 }}){
            // Periodic lattice
            // In a multi-gpu implementation these values could be loaded from a different device
            if (threadIdx.x < {{ radius_c }}){
                s_data[threadIdx.y + {{ radius_c }} ][threadIdx.x + {{ blockx1 }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdx11 }})];
            }
        }
        else {
            if (threadIdx.x < {{ radius_c }}){
                s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ blockx1 }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdx12 }}];
            }
        }
        // North halo
        if (blockIdx.y == 0){
            // Periodic lattice
            // In a multi-gpu implementation these values could be loaded from a different device
            if (threadIdx.y < {{ radius_c }}){
                s_data[threadIdx.y][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdy01 }}];
            }
        }
        else {
            if (threadIdx.y < {{ radius_c }}){
                s_data[threadIdx.y][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdy02 }})];
             }
        }
        // South halo
        if (blockIdx.y == {{ gridy1 }}){
            // Periodic lattice
            // In a multi-gpu implementation these values could be loaded from a different device
            if (threadIdx.y < {{ radius_c }}){
                s_data[threadIdx.y + {{ blocky1 }}][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx - ({{ blockIdy11 }})];
            }
        }
        else {
            if (threadIdx.y < {{ radius_c }}){
                s_data[threadIdx.y + {{ blocky1 }}][threadIdx.x + {{ radius_c }}] = u{{ tensor_ij_c }}_m[out_idx + {{ blockIdy12 }}];
             }
        }
       
	__syncthreads();


    /////////////////////////////////////////
    // Calculations

    // Discretized Laplacian operator
    // gw_coeff[0] = dt*a(t)^2/(dx^2)
    // c2_coeff's = laplacian discretization coefficients

    {% if radius_c == 4%}
        D2u = gw_coeff[0]*(c2_coeff[4]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp4 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm4 }}] +
                                       s_data[threadIdx.y + {{ radiusp4 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm4 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm1 }}] + down[{{ radiusm1 }}]) +
                          c2_coeff[3]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp3 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm3 }}] +
                                       s_data[threadIdx.y + {{ radiusp3 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm3 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm2 }}] + down[{{ radiusm2 }}]) +   
                          c2_coeff[2]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp2 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm2 }}] +
                                       s_data[threadIdx.y + {{ radiusp2 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm2 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm3 }}] + down[{{ radiusm3 }}]) +
                          c2_coeff[1]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusp1 }}] +
                                       s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radiusm1 }}] +
                                       s_data[threadIdx.y + {{ radiusp1 }}][threadIdx.x + {{ radius_c }}] +
                                       s_data[threadIdx.y + {{ radiusm1 }}][threadIdx.x + {{ radius_c }}] +
                                       up[{{ radiusm4 }}] + down[{{ radiusm4 }}]) +
                          c2_coeff[0]*(s_data[threadIdx.y + {{ radius_c }}][threadIdx.x + {{ radius_c }}]));
    {% endif %}

	/////////////////////////////////
	//  Evolution step
	/////////////////////////////////

          piu{{ tensor_ij_c }}_m[out_idx] += D2u;


    }

}


