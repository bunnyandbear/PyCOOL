#include "hip/hip_runtime.h"
/* PyCOOL v. 0.997300203937
Copyright (C) 2011/04 Jani Sainio <jani.sainio@utu.fi>
Distributed under the terms of the GNU General Public License
http://www.gnu.org/licenses/old-licenses/gpl-2.0.txt

Please cite arXiv:
if you use this code in your research.
See also http://www.physics.utu.fi/tiedostot/theory/particlecosmology/pycool .

Part of this code adapted from CUDAEASY
http://www.physics.utu.fi/tiedostot/theory/particlecosmology/cudaeasy/
(See http://arxiv.org/abs/0911.5692 for more information.),
LATTICEEASY
http://www.science.smith.edu/departments/Physics/fstaff/gfelder/latticeeasy/
and from DEFROST http://www.sfu.ca/physics/cosmology/defrost .
(See http://arxiv.org/abs/0809.4904 for more information.)
*/

__device__ void evo_step_2({{ real_name_c }} *f01{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *f0{{i}}{% endfor %}, {{ real_name_c }} *pi01{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *pi0{{i}}{% endfor %}, {{ real_name_c }} *f_lin_01_1{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *f_lin_01_{{i}}{% endfor %}, {{ real_name_c }} *pi_lin_01_1{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *pi_lin_01_{{i}}{% endfor %}, {{ real_name_c }} *f_lin_10_1{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *f_lin_10_{{i}}{% endfor %}, {{ real_name_c }} *pi_lin_10_1{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *pi_lin_10_{{i}}{% endfor %}, {{ real_name_c }} k2, {{ real_name_c }} *a, {{ real_name_c }} *b, {{ real_name_c }} *p_a, {{ real_name_c }} *t, {{ real_name_c }} deta)
// Second order symplectic integrator
// Uses pointers to register memory
{

        //{{ real_name_c }} b;


        ////////////////////////////////////
        // H1
        // Update a
        // a_coeff = -1.0/(6*V_L*m_PL^2)
        a[0] += ({{ a_coeff }})*p_a[0]*0.5*deta;

        ////////////////////////////////////
        // H2
        b[0] = 1.0/a[0];
        p_a[0] += (b[0]*b[0]*b[0]*({{ VL }})*(pi01[0]*pi01[0]{% for i in range(2,fields_c+1) %} + pi0{{i}}[0]*pi0{{i}}[0]{% endfor %}) - {{ rho_m }})*0.5*deta;

        // Background field
        {% for i in range(1,fields_c+1) %}
        f0{{i}}[0] += pi0{{i}}[0]*b[0]*b[0]*0.5*deta;{% endfor %}

        ////////////////////////////////////
        // H3

        p_a[0] += -4.0*a[0]*a[0]*a[0]*({{ V_term }})*({{ VL }})*0.5*deta;

        // Background momentum field
        {% for i in range(1,fields_c+1) %}
        pi0{{i}}[0] -= a[0]*a[0]*a[0]*a[0]*({{ dV[i-1] }})*0.5*deta;{% endfor %}

        t[0] += a[0]*0.5*deta;

        ////////////////////////////////////////////////
        // Evolve perturbations

        // Perturbation field evolution
        // For f_lin_10i initially f_lin_10i = 1.0 and pi_lin_10i = 0.0
        // and vice versa for f_lin_11i and pi_lin_11i
        {% for i in range(1,fields_c+1) %}
        f_lin_01_{{i}}[0] += pi_lin_01_{{i}}[0]*b[0]*b[0]*0.5*deta;
        f_lin_10_{{i}}[0] += pi_lin_10_{{i}}[0]*b[0]*b[0]*0.5*deta;
        {% endfor %}

        // Perturbation field evolution
        // For f_lin_10i initially f_lin_10i = 1.0 and pi_lin_10i = 0.0
        // and vice versa for f_lin_11i and pi_lin_11i

        {% for i in range(1,fields_c+1) %}
        pi_lin_01_{{i}}[0] -= a[0]*a[0]*(k2 + a[0]*a[0]*({{ d2V0[i-1] }}))*f_lin_01_{{i}}[0]*deta;
        pi_lin_10_{{i}}[0] -= a[0]*a[0]*(k2 + a[0]*a[0]*({{ d2V1[i-1] }}))*f_lin_10_{{i}}[0]*deta;
        {% endfor %}

        // Perturbation field evolution
        // For f_lin_10i initially f_lin_10i = 1.0 and pi_lin_10i = 0.0
        // and vice versa for f_lin_11i and pi_lin_11i

        {% for i in range(1,fields_c+1) %}
        f_lin_01_{{i}}[0] += pi_lin_01_{{i}}[0]*b[0]*b[0]*0.5*deta;
        f_lin_10_{{i}}[0] += pi_lin_10_{{i}}[0]*b[0]*b[0]*0.5*deta;
        {% endfor %}

        ////////////////////////////////////
        // H3

        p_a[0] += -4.0*a[0]*a[0]*a[0]*({{ V_term }})*({{ VL }})*0.5*deta;

        // Background momentum field
        {% for i in range(1,fields_c+1) %}
        pi0{{i}}[0] -= a[0]*a[0]*a[0]*a[0]*({{ dV[i-1] }})*0.5*deta;{% endfor %}

        t[0] += a[0]*0.5*deta;
        ////////////////////////////////////
        // H2

        p_a[0] += (b[0]*b[0]*b[0]*({{ VL }})*(pi01[0]*pi01[0]{% for i in range(2,fields_c+1) %} + pi0{{i}}[0]*pi0{{i}}[0]{% endfor %}) - {{ rho_m }})*0.5*deta;

        // Background field
        {% for i in range(1,fields_c+1) %}
        f0{{i}}[0] += pi0{{i}}[0]*b[0]*b[0]*0.5*deta;{% endfor %}

        ////////////////////////////////////
        // H1
        // Update a
        a[0] += {{ a_coeff }}*p_a[0]*0.5*deta;

}



//////////////////////////////////////////////////////////////////////
// Linearized perturbation evolution solver
//////////////////////////////////////////////////////////////////////
// This kernel is used to evolve linearized equations with two different initial values:
// for dfield_i_10 and dpi_i_10_m initially dfield = 1 and dpi_i = 0
// and similarly for dfield_i_01 and dpi_i_01_m initially dfield = 0 and dpi_i = 1.

__global__ void linear_evo({{ real_name_c }} *f01_m{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *f0{{i}}_m{% endfor %}, {{ real_name_c }} *pi01_m{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *pi0{{i}}_m{% endfor %}, {{ real_name_c }} *f_lin_01_1_m{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *f_lin_01_{{i}}_m{% endfor %}, {{ real_name_c }} *pi_lin_01_1_m{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *pi_lin_01_{{i}}_m{% endfor %}, {{ real_name_c }} *f_lin_10_1_m{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *f_lin_10_{{i}}_m{% endfor %}, {{ real_name_c }} *pi_lin_10_1_m{% for i in range(2,fields_c+1) %}, {{ real_name_c }} *pi_lin_10_{{i}}_m{% endfor %}, {{ real_name_c }} *a_val, {{ real_name_c }} *p_a_val, {{ real_name_c }} *t_val, {{ real_name_c }} deta, int steps, {{ real_name_c }} *k2_bins)

{

    volatile unsigned int in_idx = blockIdx.x*blockDim.x+threadIdx.x;

    volatile unsigned int i;
    {{ real_name_c }} a, p_a, k2, t;
    {{ real_name_c }} b;

    {% for i in range(1,fields_c+1) %}
    {{ real_name_c }} f0{{i}};{% endfor %}
    {% for i in range(1,fields_c+1) %}
    {{ real_name_c }} pi0{{i}};{% endfor %}
    {% for i in range(1,fields_c+1) %}
    {{ real_name_c }} f_lin_01_{{i}};{% endfor %}
    {% for i in range(1,fields_c+1) %}
    {{ real_name_c }} f_lin_10_{{i}};{% endfor %}
    {% for i in range(1,fields_c+1) %}
    {{ real_name_c }} pi_lin_01_{{i}};{% endfor %}
    {% for i in range(1,fields_c+1) %}
    {{ real_name_c }} pi_lin_10_{{i}};{% endfor %}

    // Include only non-homogeneous (k != 0) modes:
    k2 = k2_bins[in_idx];

    // Set initial values

    {% for i in range(1,fields_c+1) %}
    f0{{i}} = f0{{i}}_m[0];
    pi0{{i}} = pi0{{i}}_m[0];

    f_lin_01_{{i}} = 0.0;
    f_lin_10_{{i}} = 1.0;

    pi_lin_01_{{i}} = 1.0;
    pi_lin_10_{{i}} = 0.0;
    {% endfor %}

    a = a_val[0];
    p_a = p_a_val[0];
    t = t_val[0];


    __syncthreads();

    /////////////////////////////////////////
    // Calculations

    /////////////////////////////////
    //  Evolution step
    /////////////////////////////////

    for(i=0; i<(steps); i++)
    {
    {% if order == 2 %}
        evo_step_2(&f01{% for i in range(2,fields_c+1) %}, &f0{{i}}{% endfor %}, &pi01{% for i in range(2,fields_c+1) %}, &pi0{{i}}{% endfor %}, &f_lin_01_1{% for i in range(2,fields_c+1) %}, &f_lin_01_{{i}}{% endfor %}, &pi_lin_01_1{% for i in range(2,fields_c+1) %}, &pi_lin_01_{{i}}{% endfor %}, &f_lin_10_1{% for i in range(2,fields_c+1) %}, &f_lin_10_{{i}}{% endfor %}, &pi_lin_10_1{% for i in range(2,fields_c+1) %}, &pi_lin_10_{{i}}{% endfor %}, k2, &a, &b, &p_a, &t, deta);
    {% else %}
    {% for w in w_i %}
        evo_step_2(&f01{% for i in range(2,fields_c+1) %}, &f0{{i}}{% endfor %}, &pi01{% for i in range(2,fields_c+1) %}, &pi0{{i}}{% endfor %}, &f_lin_01_1{% for i in range(2,fields_c+1) %}, &f_lin_01_{{i}}{% endfor %}, &pi_lin_01_1{% for i in range(2,fields_c+1) %}, &pi_lin_01_{{i}}{% endfor %}, &f_lin_10_1{% for i in range(2,fields_c+1) %}, &f_lin_10_{{i}}{% endfor %}, &pi_lin_10_1{% for i in range(2,fields_c+1) %}, &pi_lin_10_{{i}}{% endfor %}, k2, &a, &b, &p_a, &t, {{w}}*deta);{% endfor %}
    {%endif %}

    }

    // Write final values
    // Commented writes were used for debugging
    {% for i in range(1,fields_c+1) %}

    f_lin_01_{{i}}_m[in_idx] = f_lin_01_{{i}};
    f_lin_10_{{i}}_m[in_idx] = f_lin_10_{{i}};

    pi_lin_01_{{i}}_m[in_idx] = pi_lin_01_{{i}};
    pi_lin_10_{{i}}_m[in_idx] = pi_lin_10_{{i}};
    {% endfor %}

    //k2_field[in_idx] = k2;

    if((threadIdx.x==0)&&(threadIdx.y==0)&&(blockIdx.x==0)&&(blockIdx.y==0))
    {
        a_val[0] = a;
        p_a_val[0] = p_a;
        t_val[0] = t;

        {% for i in range(1,fields_c+1) %}
        f0{{i}}_m[0] = f0{{i}};
        pi0{{i}}_m[0] = pi0{{i}};
        {% endfor %}
    }

}

